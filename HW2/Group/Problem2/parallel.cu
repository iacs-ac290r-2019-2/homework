#include "hip/hip_runtime.h"
/* Daniel Willen, 2019
 *
 * Solve the transient heat conduction problem with homogeneous Dirichlet
 *  boundary conditions:
 *
 *    u(x={0,L}) = u(y={0,L}) = 0
 *
 *  and initial condition:
 *
 *    u(x,y,0) = sin(x) * sin(y)
 *
 *  on the domain 0 <= x,y <= L, with L = pi.
 *
 * This program solves the above problem on a single GPU with the Jacobi method.
 * 
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#define PI 3.14159265358979323846
#define MAX_THREADS_DIM 18        // Note that this depends on the hardware

/* Note on the structure of this file:
 *  - Cuda device constant memory declarations are at the top
 *  - Functions definitions are in the middle. Functions include:
 *  - - parse_cmdline: Read command-line arguments for domain size
 *  - - jacobi_solver: Advance the soln to the next time step using Jacobi
 *  - - check_error:   Calculate the error b/t the numeric and analytic solns
 *  - The `main' function is at the bottom
 *
 *  Note that it is good practice to use header files and break functions out
 *   into separate files. This has not been done here for simplicity.
 */

/*** Auxiliary Functions ***/

/* Read the command line inputs */
// - argv[0] is the program name
// - argv[1] is the first input (number of points)
int parse_cmdline(int argc, char *argv[]) {
  int nx;
  if (argc >= 2) {
    nx = atoi(argv[1]); // Number of grid points
    printf("Grid is %d by %d\n\n", nx, nx);
  } else {
    printf("Input error. Run like: \n\n");
    printf("  $ ./parallel.c n\n\n");
    printf("  where n is the number of grid cells in one dimension\n");
    exit(EXIT_FAILURE);
  }
  return nx;
}

/*** GPU Constants ***/
__constant__ int _nx;
__constant__ int _ny;
__constant__ double _Lx;
__constant__ double _Ly;
__constant__ double _dx;
__constant__ double _dy;
__constant__ double _dt;
__constant__ double _D;
__constant__ double _pref;

/*******************************************************************************
 * Step IV: Launch the GPU kernel to advance to the next time step with the    *
 *          Jacobi method here.                                                *
 ******************************************************************************/

__global__ void jacobi_solver(double* u, double* u_new) {

  // int ti = blockIdx.x*blockDim.x + threadIdx.x;
  // int tj = blockIdx.y*blockDim.y + threadIdx.y;

  // if ((ti >= 1 && ti < (_nx-1)) && 
  //     (tj >= 1 && tj < (_ny-1))) {
  //   u_new[ti + tj*_nx] = 
  //       u[ti + tj*_nx] + _pref * (
  //         u[(ti+1) + tj*_nx] + 
  //         u[(ti-1) + tj*_nx] + 
  //         u[ti + (tj+1)*_nx] + 
  //         u[ti + (tj-1)*_nx] + 
  //         u[ti + tj*_nx] * (-4)
  //       );
  // }

  __shared__ double s_u[MAX_THREADS_DIM*MAX_THREADS_DIM];

  int si = threadIdx.x;
  int sj = threadIdx.y;
  int ti = blockIdx.x*(blockDim.x-2) + threadIdx.x;
  int tj = blockIdx.y*(blockDim.y-2) + threadIdx.y;
  
  if (ti < _nx && tj < _ny) {
    s_u[si + sj*blockDim.x] = u[ti + tj*_nx];
  }

  __syncthreads();

  if ((ti >= 1 && ti < (_nx-1)) && 
      (tj >= 1 && tj < (_ny-1)) &&
      (si > 0 && si < (blockDim.x-1)) &&
      (sj > 0 && sj < (blockDim.y-1))) {
    u_new[ti + tj*_nx] = 
        s_u[si + sj*blockDim.x] + _pref * (
          s_u[(si+1) + sj*blockDim.x] + 
          s_u[(si-1) + sj*blockDim.x] + 
          s_u[si + (sj+1)*blockDim.x] + 
          s_u[si + (sj-1)*blockDim.x] + 
          s_u[si + sj*blockDim.x] * (-4)
        );
  }

  return;
}

/******************************************************************************
 * Step V: Launch the GPU kernel to calculate the error at each grid point    *
 *         here.                                                              *
 *****************************************************************************/

__global__ void check_error(double* u, double* error, double time) {

  // int ti = blockIdx.x*blockDim.x + threadIdx.x;
  // int tj = blockIdx.y*blockDim.y + threadIdx.y;

  // if ((ti >= 1 && ti < (_nx-1)) && 
  //     (tj >= 1 && tj < (_ny-1))) {
  //   error[ti + tj*_nx] = u[ti + tj*_nx] / (sin(ti*_dx) * sin(tj*_dy) * exp(-2*_D*time)) - 1;
  // }

  int ti = blockIdx.x*(blockDim.x-2) + threadIdx.x;
  int tj = blockIdx.y*(blockDim.y-2) + threadIdx.y;

  if ((ti >= 1 && ti < (_nx-1)) && 
      (tj >= 1 && tj < (_ny-1)) &&
      (threadIdx.x > 0 && threadIdx.x < (blockDim.x-1)) &&
      (threadIdx.y > 0 && threadIdx.y < (blockDim.y-1))) {
    error[ti + tj*_nx] = u[ti + tj*_nx] / (sin(ti*_dx) * sin(tj*_dy) * exp(-2*_D*time)) - 1;
  }

  return;
}

/*** Main Function ***/
int main(int argc, char *argv[])
{
  /* Variable declaration */
  double Lx = PI;           // Domain length in x-direction
  double Ly = PI;           // Domain length in y-direction
  double D = 1.;            // Diffusion constant

  int nx, ny;               // Grid points (grid cells + 1)
  double dx, dy;            // Grid spacing
  double dt;                // Time step size
  double sim_time;          // Length of sim time, arbitrary for simplicity
  double pref;              // Pre-factor in the Jacobi method

  double error = 0.;        // Mean percent-difference at each grid point
  error = error;            // To prevent compiler warning

  /* Parse command-line for problem size */
  nx = parse_cmdline(argc, argv);
  ny = nx;                  // Assume a square grid

  /* Initialize variables */
  dx = Lx / (nx - 1);       // Cell width in x-direction
  dy = Ly / (ny - 1);       // Cell width in y-direction
  dt = 0.25*dx*dy/D;        // Limited by diffusive stability
  sim_time = Lx*Ly/D;       // Arbitrary simulation length
  pref = D*dt/(dx*dx);      // Jacobi pre-factor

  printf("Parameters\n");
  printf("---------------------------\n");
  printf("Lx = %.5lf\n", Lx); 
  printf("Lx = %.5lf\n", Ly); 
  printf("T  = %.5lf\n", sim_time); 
  printf("D  = %.5lf\n", D);
  printf("nx = %d\n", nx);
  printf("ny = %d\n", nx);
  printf("dx = %.5lf\n", dx);
  printf("dy = %.5lf\n", dy);
  printf("dt = %.5lf\n", dt);
  printf("\n");

  hipMemcpyToSymbol(HIP_SYMBOL(_nx), &nx, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_ny), &ny, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_Lx), &Lx, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_Ly), &Ly, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_dx), &dx, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_dy), &dy, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_dt), &dt, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_D),  &D,  sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_pref), &pref, sizeof(double));

  /*****************************************************************************
   * Step I: Declare, allocate, and initialize memory for the field variable   *
   *         u on the CPU.                                                     *
   ****************************************************************************/

  double *u;
  u = (double*) malloc(nx*ny * sizeof(double));
  for (int i = 0; i < nx; i++) {
    for (int j = 0; j < ny; j++) {
      u[i+j*nx] = sin(i*dx) * sin(j*dy);
    }
  }

  /*****************************************************************************
   * Step II: Declare and allocate GPU memory for _u, _u_new, and _error. Copy *
   *          the initial condition to the GPU.                                *
   ****************************************************************************/

  double *_u, *_u_new, *_error;
  hipMalloc(&_u, nx*ny * sizeof(double));
  hipMalloc(&_u_new, nx*ny * sizeof(double));
  hipMalloc(&_error, nx*ny * sizeof(double));
  hipMemcpy(_u, u, nx*ny * sizeof(double), hipMemcpyHostToDevice);

  // Set the new soln and error to 0
  hipMemset(_u_new, 0., nx*ny * sizeof(double));
  hipMemset(_error, 0., nx*ny * sizeof(double));

  // Create thrust pointers to device memory for error calculation
  thrust::device_ptr<double> t_error(_error);

  /*****************************************************************************
   * Step III: Set up the kernel execution configuration for the domain based  *
   *           on the input domain size and the MAX_THREADS_DIM variable.      *
   ****************************************************************************/

  int threads_x = MAX_THREADS_DIM;
  int threads_y = MAX_THREADS_DIM;

  // int blocks_x = (int) ceil((double) nx / (double) threads_x);
  // int blocks_y = (int) ceil((double) ny / (double) threads_y);
  int blocks_x = (int) ceil((double) nx / (double) (threads_x - 2));
  int blocks_y = (int) ceil((double) ny / (double) (threads_y - 2));

  dim3 dim_blocks(threads_x, threads_y);
  dim3 num_blocks(blocks_x, blocks_y);

  printf("Parallelization\n");
  printf("---------------------------\n");
  printf("MAX_THREADS_DIM = %d\n", MAX_THREADS_DIM);
  printf("threads_x = %d\n", threads_x); 
  printf("threads_y = %d\n", threads_y); 
  printf("blocks_x  = %d\n", blocks_x); 
  printf("blocks_y  = %d\n", blocks_y); 
  printf("\n");

  /***************************/
  /* Main Time-Stepping Loop */
  /***************************/

  for (double time = 0.; time <= sim_time; time += dt) {
    /***************************************************************************
     * Step IV: Launch the GPU kernel to advance to the next time step with    *
     *          the Jacobi method here.                                        *
     **************************************************************************/

    jacobi_solver<<<num_blocks, dim_blocks>>>(_u, _u_new);

    /***************************************************************************
     * Step V: Launch the GPU kernel to calculate the error at each grid point *
     *         here.                                                           *
     **************************************************************************/

    check_error<<<num_blocks, dim_blocks>>>(_u, _error, time);

    // Use thrust to do a parallel reduction on the error
    error = thrust::reduce(t_error, t_error + nx*ny, 0., thrust::plus<double>());
    printf("Error at t* = %.5lf is %e\n", time*D/(Lx*Lx), error/(nx*ny));

    // Copy new soln to old. This also blocks to ensure computations are finished.
    hipMemcpy(_u, _u_new, nx*ny * sizeof(double), hipMemcpyDeviceToDevice);
  }

  /*****************************************************************************
   * Step VI: Copy the memory back to the CPU.                                 *
   ****************************************************************************/

  hipMemcpy(u, _u, nx*ny * sizeof(double), hipMemcpyDeviceToHost);

  /*****************************************************************************
   * Step I and Step II: Free the memory that you declared and allocated       *
   *                     earlier in the program.                               *
   ****************************************************************************/

  hipFree(_u);
  hipFree(_u_new);
  hipFree(_error);
  free(u);

  return EXIT_SUCCESS;
}

